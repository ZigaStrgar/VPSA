
#include "common/book.h"

int main(void){
    hipDeviceProp_t devprop;
    int count;
    
    hipGetDeviceCount(&count);
    
    for (int i = 0; i < count; i++) {
        hipGetDeviceProperties(&devprop, i);
        
        printf("  ------ Info o CUDA napravi %d ------  ", i);
        printf("Ime naprave: %s \n", devprop.name);
        printf("Hitrost ure: %d \n", devprop.clockRate);
        printf("Kolicina globalnega pomnilnika: %d \n", devprop.totalGlobalMem);
        printf("Stevilo multiprocesorjev SM: %d\n", devprop.multiProcessorCount);
        printf("Kolicina skupnega pomnilnika v SM: %d \n", devprop.sharedMemPerMultiprocessor);
        printf("Stevilo registrov v SM: %d \n", devprop.regsPerMultiprocessor);
        
        printf("Max stevilo niti v bloku: %d \n", devprop.maxThreadsPerBlock);
        printf("Stevilo registrov na blok: %d \n", devprop.regsPerBlock);
        printf("Kolicina skupnega pomnilnika za blok: %d \n", devprop.sharedMemPerBlock);
        printf("Max stevilo niti na SM: %d \n", devprop.maxThreadsPerMultiProcessor);
        printf("Velikost snopa (warp): %d\n", devprop.warpSize);
    }
}
