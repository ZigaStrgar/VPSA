#include "hip/hip_runtime.h"
#include <stdio.h>
#include "common/book.h"

#define N 10

__global__ void vectorAddGPU(float *a, float *b, float *c){
    int tid = threadIdx.x;
    
    if(tid < N){
        c[tid] = a[tid] + b[tid];
    }
}

void vectorAddSerial(float *a, float *b, float *c){
    int tid = 0;
    
    while(tid < N){
        c[tid] = a[tid] + b[tid];
        tid++;
    }
}

int main(void){
    float *h_a, *h_b, *h_c;
    float *d_a, *d_b, *d_c;
    
    h_a = (float*) malloc(N*sizeof(float));
    h_b = (float*) malloc(N*sizeof(float));
    h_c = (float*) malloc(N*sizeof(float));
    
    hipMalloc( (void**) &d_a, N*sizeof(float));
    hipMalloc( (void**) &d_b, N*sizeof(float));
    hipMalloc( (void**) &d_c, N*sizeof(float));
    
    for(int i = 0; i < N; i++){
        h_a[i] = (float)-i;
        h_b[i] = (float)(i*i);
    }
    
    hipMemcpy(d_a, h_a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * sizeof(float), hipMemcpyHostToDevice);
    
    //vectorAddSerial(a,b,c);
    vectorAddGPU<<<1, N>>>(d_a, d_b, d_c);
    
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("%s \n", hipGetErrorString(err));
    }
    
    hipDeviceSynchronize();
    
    hipMemcpy(h_c, d_c, N*sizeof(float), hipMemcpyDeviceToHost);
    
    for (int i = 0; i < N; i++) {
        printf("%f + %f = %f \n", h_a[i], h_b[i], h_c[i]);
    }
    
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    
    free(h_a);
    free(h_b);
    free(h_c);
    
    return 0;
}
